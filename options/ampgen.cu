#include "hip/hip_runtime.h"
#include <nppdefs.h>
#include <iostream>
#include <math.h>
#include <array>
#include <hip/hip_complex.h>

#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>

namespace ampgen_cuda {
  struct complex_t {
    hipFloatComplex v;
    __host__ __device__ complex_t( const hipFloatComplex& v ) : v(v) {}
    __host__ __device__ complex_t( const float_t& r, const float_t& i=0 ) : v(make_hipFloatComplex(r,i)) {}
    __host__ __device__ float real() const { return hipCrealf(v) ; } 
    __host__ __device__ float imag() const { return hipCimagf(v) ; } 
  };
  __host__ __device__ __inline__ complex_t operator+( complex_t a, complex_t b ){ return hipCaddf(a.v,b.v) ; } 
  __host__ __device__ __inline__ complex_t operator-( complex_t a, complex_t b ){ return hipCsubf(a.v,b.v) ; } 
  __host__ __device__ __inline__ complex_t operator/( complex_t a, complex_t b ){ return hipCdivf(a.v,b.v) ; } 
  __host__ __device__ __inline__ complex_t operator*( complex_t a, complex_t b ){ return hipCmulf(a.v,b.v) ; } 
  
  __host__ __device__ __inline__ complex_t operator+( float a, complex_t b ){ return complex_t( a + b.v.x, b.v.y ) ; }
  __host__ __device__ __inline__ complex_t operator-( float a, complex_t b ){ return complex_t( a - b.v.x, -b.v.y) ; } 
  __host__ __device__ __inline__ complex_t operator/( float a, complex_t b ){ float p = a / ( b.v.x*b.v.x + b.v.y*b.v.y) ; return complex_t( p * b.v.x, -p*b.v.y); } 
  __host__ __device__ __inline__ complex_t operator*( float a, complex_t b ){ return complex_t( a * b.v.x, a*b.v.y) ; } 
  
  __host__ __device__ __inline__ complex_t operator+( complex_t a, float b ){ return complex_t( a.v.x + b , a.v.y  ) ; } 
  __host__ __device__ __inline__ complex_t operator-( complex_t a, float b ){ return complex_t( a.v.x -b  , a.v.y ); } 
  __host__ __device__ __inline__ complex_t operator/( complex_t a, float b ){ return complex_t( a.v.x /b  , a.v.y/b ) ; } 
  __host__ __device__ __inline__ complex_t operator*( complex_t a, float b ){ return complex_t( b * a.v.x, b * a.v.y ) ; } 
  __host__ __inline__ std::ostream& operator<<( std::ostream& l, complex_t b ){ return l << "(" << b.real() << "," << b.imag() << ")"; }
};
using namespace ampgen_cuda; 

__global__ void bw_kernel( complex_t* __restrict__ r, const int N, const float* __restrict__ x0, const float3* __restrict__ x1)
{
  int i     = blockIdx.x * blockDim.x + threadIdx.x;
  float v899190686 = (sqrt((0.243717) + x1[i].x*x1[i].x + x1[i].y*x1[i].y + x1[i].z*x1[i].z) + sqrt((0.01948) + x1[i+N].x*x1[i+N].x + x1[i+N].y*x1[i+N].y + x1[i+N].z*x1[i+N].z))*(sqrt((0.243717) + x1[i].x*x1[i].x + x1[i].y*x1[i].y + x1[i].z*x1[i].z) + sqrt((0.01948) + x1[i+N].x*x1[i+N].x + x1[i+N].y*x1[i+N].y + x1[i+N].z*x1[i+N].z))-(x1[i].z + x1[i+N].z)*(x1[i].z + x1[i+N].z)-(x1[i].y + x1[i+N].y)*(x1[i].y + x1[i+N].y)-(x1[i].x + x1[i+N].x)*(x1[i].x + x1[i+N].x);
  float v2885272051 = (0.25)*v899190686-(0.131598) + (0.012571)/v899190686;
  r[i] = sqrt((0.900316)*x0[1]*x0[2]*x0[1]*sqrt(x0[2]*x0[2] + x0[1]*x0[1])/sqrt(x0[1]*sqrt(x0[2]*x0[2] + x0[1]*x0[1]) + x0[1]*x0[1]))*sqrt((9.)/((9.) + (3.)*v2885272051*x0[3]*x0[3] + v2885272051*x0[3]*x0[3]*v2885272051*x0[3]*x0[3]))/(x0[1]*x0[1]-v899190686-ampgen_cuda::complex_t(0.,1.)*x0[1]*x0[2]*((9.) + (3.)*x0[3]*((0.25)*x0[1]*x0[1]-(0.131598) + (0.050282)/((4.)*x0[1]*x0[1]))*x0[3] + x0[3]*((0.25)*x0[1]*x0[1]-(0.131598) + (0.050282)/((4.)*x0[1]*x0[1]))*x0[3]*x0[3]*((0.25)*x0[1]*x0[1]-(0.131598) + (0.050282)/((4.)*x0[1]*x0[1]))*x0[3])*sqrt(v2885272051/((0.25)*x0[1]*x0[1]-(0.131598) + (0.050282)/((4.)*x0[1]*x0[1])))*v2885272051*v2885272051*x0[1]*rsqrt(v899190686)/((0.25)*x0[1]*x0[1]-(0.131598) + (0.050282)/((4.)*x0[1]*x0[1]))/((0.25)*x0[1]*x0[1]-(0.131598) + (0.050282)/((4.)*x0[1]*x0[1]))/((9.) + (3.)*v2885272051*x0[3]*x0[3] + v2885272051*x0[3]*x0[3]*v2885272051*x0[3]*x0[3]));
}

#include "output.h"



std::vector<std::string> split( const std::string& s, char delim, bool ignoreWhitespace=true )
{
  std::vector<std::string> elems;
  std::string item;
  std::stringstream ss( s );
  while ( std::getline( ss, item, delim ) ) {
    if ( !ignoreWhitespace || ( item != " " && item != "" && item != "\n" && item != "\t" ) ) elems.push_back( item );
  }
  return elems;
}

int main(void)
{
  int N = 1 << 23; 
  complex_t * r;
  float     * pHost; 
  float3    * xE;
  std::ifstream stream("events.dat");
  std::string tmp;
  std::cout << sizeof(float3) << " " << sizeof(float) << std::endl; 
    
  hipMallocManaged( &r    , sizeof(complex_t) * N  );
  hipMallocManaged( &xE   , sizeof(float3)    * N * 3 );
  hipMallocManaged( &pHost, sizeof(float)     * 4 );
  pHost[0] = 5 ;
  pHost[1] = 1.4324;
  pHost[2] = 0.109;
  pHost[3] = 1.5;

  std::vector<float> event_full( 12 * N );
  std::getline( stream , tmp );
  for (int i = 0; i < N+1; i++) {
    r[i]  = complex_t(0.0f,0.0f);
    std::getline( stream, tmp );
    auto tokens = split( tmp, ',');
      if( i== 0 ) std::cout << tmp << std::endl; 
    for( int p = 0 ; p < 3 ; ++p ){
      xE[ i + N * p  ].x  = stof( tokens[4*p]   );
      xE[ i + N * p  ].y  = stof( tokens[4*p+1] );
      xE[ i + N * p  ].z  = stof( tokens[4*p+2] );
    }
    for( int j = 0 ; j < 12; ++j ) 
      event_full[ j + 12*i ] = stof( tokens[j] );
  }
  for( int i = 0 ; i < 100 ; ++i )   bw_kernel<<< N/128, 128 >>>(r, N, pHost, xE );
  for( int i = 0 ; i < 100 ; ++i )   p2540321052 <<< N/128, 128 >>>(r, N, pHost, xE );

  hipDeviceSynchronize();

  std::ofstream output("events_out.dat");
  for( int i = 0 ; i < N ; ++i )
  {
    for( int j = 0 ; j < 12 ; ++j ) output << event_full[j + 12*i] <<  " ";
    output << r[i] << std::endl;  
  }
  output.close();
  hipFree(r);
  
  hipFree(xE);
  hipFree( pHost );
  return 0;
  
}
